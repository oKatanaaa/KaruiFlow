﻿#include "SoftmaxCUDAKernels.h"
#include "../../core/headers/CudaContextManager.h"
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include ""


namespace karuiflow {
	void SoftmaxCudaKernel::forward(std::vector<Storage*> inputs, Storage* output) {
		// Cuda kernels are guarantied to receive Storages that store their data
		// on device (cuda device).
		float* inputData = (float*)inputs[0]->getData();
		float* outputData = (float*)output->getData();
		hipdnnHandle_t* handle = CudaContextManager::getCudnnHandle();

		std::vector<int> inputDataShape = inputs[0]->getShape();
		// For 2D Tensor: n=batch-size, c=channels
		int n = inputDataShape[0]; 
		int c = inputDataShape[1];
		int h = 1, w = 1;
		hipdnnTensorDescriptor_t inputDesc;
		hipdnnCreateTensorDescriptor(&inputDesc);
		hipdnnSetTensor4dDescriptor(inputDesc,
			HIPDNN_TENSOR_NCHW,
			HIPDNN_DATA_FLOAT,
			n, c, h, w);

		hipdnnTensorDescriptor_t outputDesc;
		hipdnnCreateTensorDescriptor(&outputDesc);
		hipdnnSetTensor4dDescriptor(outputDesc,
			HIPDNN_TENSOR_NCHW,
			HIPDNN_DATA_FLOAT,
			n, c, h, w);

		const float alpha = 1, beta = 0;
		// Straightforward softmax operation is computed 
		// per spatial location (H,W) per image (N) across dimension C.
		hipdnnSoftmaxForward(*handle,
			HIPDNN_SOFTMAX_FAST, HIPDNN_SOFTMAX_MODE_CHANNEL,
			&alpha, inputDesc, inputData,
			&beta,outputDesc, outputData);
	}

	void SoftmaxCudaKernel::backward(std::vector<Storage*> inputs, std::vector<bool> requiresGrad,
		Storage* outerGradient, std::vector<Storage*> outputGradients) {
		if (requiresGrad[0]) {
			hipdnnHandle_t* handle = CudaContextManager::getCudnnHandle();
			std::vector<int> inputDataShape = inputs[0]->getShape();
			// For 2D Tensor: n=batch-size, c=channels
			int n = inputDataShape[0];
			int c = inputDataShape[1];
			int h = 1, w = 1;
			hipdnnTensorDescriptor_t inputDesc;
			hipdnnCreateTensorDescriptor(&inputDesc);
			hipdnnSetTensor4dDescriptor(inputDesc,
				HIPDNN_TENSOR_NCHW,
				HIPDNN_DATA_FLOAT,
				n, c, h, w);

			hipdnnTensorDescriptor_t outputDesc;
			hipdnnCreateTensorDescriptor(&outputDesc);
			hipdnnSetTensor4dDescriptor(outputDesc,
				HIPDNN_TENSOR_NCHW,
				HIPDNN_DATA_FLOAT,
				n, c, h, w);

			hipdnnTensorDescriptor_t gradDesc;
			hipdnnCreateTensorDescriptor(&gradDesc);
			hipdnnSetTensor4dDescriptor(gradDesc,
				HIPDNN_TENSOR_NCHW,
				HIPDNN_DATA_FLOAT,
				n, c, h, w);

			forward(inputs, outputGradients[0]);
			float* _outerGradient = (float*)outerGradient->getData();
			float* outputGradient = (float*)outputGradients[0]->getData();
			const float alpha = 1, beta = 0;
	
			hipdnnSoftmaxBackward(*handle,
				HIPDNN_SOFTMAX_FAST, HIPDNN_SOFTMAX_MODE_CHANNEL,
				&alpha, inputDesc, outputGradient,
				gradDesc, _outerGradient,
				&beta, outputDesc, outputGradient);
		}
		
	}
}
